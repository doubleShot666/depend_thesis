#include "hip/hip_runtime.h"
#include "OptionDataStruct.h"

__global__ void blackscholes_skepu_cuda_precompiled_MapKernel_mapFunction(float* skepu_output, OptionData_ *data_e,  size_t skepu_w2, size_t skepu_w3, size_t skepu_w4, size_t skepu_n, size_t skepu_base)
{
	size_t skepu_i = blockIdx.x * blockDim.x + threadIdx.x;
	size_t skepu_gridSize = blockDim.x * gridDim.x;
	

	while (skepu_i < skepu_n)
	{
		
		
		auto skepu_res = skepu_userfunction_skepu_skel_0map_mapFunction::CU(data_e[skepu_i]);
		skepu_output[skepu_i] = skepu_res;
		skepu_i += skepu_gridSize;
	}
}
