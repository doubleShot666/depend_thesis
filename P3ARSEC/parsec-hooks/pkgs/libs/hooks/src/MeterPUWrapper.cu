//
// Created by Administrator on 3/15/2021.
//
#include <MeterPU.h>
#include "MeterPUWrapper.h"
using namespace MeterPU;

/*extern "C"*/ void* MeterPU_PCM_Energy_create() {
    return new Meter<PCM_Energy>;
}
/*extern "C"*/ void MeterPU_PCM_Energy_release(void* myclass) {
    delete static_cast<Meter<PCM_Energy>*>(myclass);
}

/*extern "C"*/ void MeterPU_PCM_Energy_start(void* myclass) {
    static_cast<Meter<PCM_Energy>*>(myclass)->start();
}

/*extern "C"*/ void MeterPU_PCM_Energy_stop(void* myclass) {
    static_cast<Meter<PCM_Energy>*>(myclass)->stop();
}

/*extern "C"*/ void MeterPU_PCM_Energy_calc(void* myclass) {
    static_cast<Meter<PCM_Energy>*>(myclass)->calc();
}

/*extern "C"*/ double MeterPU_PCM_Energy_getValue(void* myclass) {
    return static_cast<Meter<PCM_Energy>*>(myclass)->get_value();
}



/*extern "C"*/ void* MeterPU_NVML_Energy_create() {
    return new Meter<NVML_Energy<>>;
}
/*extern "C"*/ void MeterPU_NVML_Energy_release(void* myclass) {
    delete static_cast<Meter<NVML_Energy<>>*>(myclass);
}

/*extern "C"*/ void MeterPU_NVML_Energy_start(void* myclass) {
    static_cast<Meter<NVML_Energy<>>*>(myclass)->start();
}

/*extern "C"*/ void MeterPU_NVML_Energy_stop(void* myclass) {
    static_cast<Meter<NVML_Energy<>>*>(myclass)->stop();
}

/*extern "C"*/ void MeterPU_NVML_Energy_calc(void* myclass) {
    static_cast<Meter<NVML_Energy<>>*>(myclass)->calc();
}

/*extern "C"*/ double MeterPU_NVML_Energy_getValue(void* myclass) {
    return static_cast<Meter<NVML_Energy<>>*>(myclass)->get_value();
}



/*extern "C"*/ void* MeterPU_CUDA_Time_create() {
    return new Meter<CUDA_Time>;
}
/*extern "C"*/ void MeterPU_CUDA_Time_release(void* myclass) {
    delete static_cast<Meter<CUDA_Time>*>(myclass);
}

/*extern "C"*/ void MeterPU_CUDA_Time_start(void* myclass) {
    static_cast<Meter<CUDA_Time>*>(myclass)->start();
}

/*extern "C"*/ void MeterPU_CUDA_Time_stop(void* myclass) {
    static_cast<Meter<CUDA_Time>*>(myclass)->stop();
}

/*extern "C"*/ void MeterPU_CUDA_Time_calc(void* myclass) {
    static_cast<Meter<CUDA_Time>*>(myclass)->calc();
}

/*extern "C"*/ double MeterPU_CUDA_Time_getValue(void* myclass) {
    return static_cast<Meter<CUDA_Time>*>(myclass)->get_value();
}



